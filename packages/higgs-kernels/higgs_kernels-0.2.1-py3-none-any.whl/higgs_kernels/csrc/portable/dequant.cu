#include "hip/hip_runtime.h"
#include <cstdint>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAException.h>

static __global__ void higgs_dequantize_2_256_ptr_cuda_portable_kernel(
	const uint8_t* __restrict__ x,
	const uint32_t* __restrict__ grid_packed,
	uint32_t* __restrict__ out_packed,
	long long out_dim) {
	__shared__ uint32_t s_grid[256];

	for (int idx = threadIdx.x; idx < 256; idx += blockDim.x) {
		s_grid[idx] = grid_packed[idx];
	}
	__syncthreads();

	long long i = static_cast<long long>(blockIdx.x) * blockDim.x + threadIdx.x;
	if (i >= out_dim) return;

	uint8_t code = x[i];
	out_packed[i] = s_grid[code];
}

extern "C" void higgs_dequantize_2_256_ptr_cuda_portable(
	uint64_t x_ptr,
	uint64_t grid_ptr,
	uint64_t out_ptr,
	int64_t out_dim) {
	const uint8_t* x = reinterpret_cast<const uint8_t*>(x_ptr);
	const uint32_t* grid_packed = reinterpret_cast<const uint32_t*>(grid_ptr);
	uint32_t* out_packed = reinterpret_cast<uint32_t*>(out_ptr);

	constexpr int threads_per_block = 256;
	int blocks = static_cast<int>((out_dim + threads_per_block - 1) / threads_per_block);

	auto stream = at::cuda::getCurrentCUDAStream();
	higgs_dequantize_2_256_ptr_cuda_portable_kernel<<<blocks, threads_per_block, 0, stream.stream()>>>(
		x, grid_packed, out_packed, static_cast<long long>(out_dim));

	C10_CUDA_KERNEL_LAUNCH_CHECK();
}
