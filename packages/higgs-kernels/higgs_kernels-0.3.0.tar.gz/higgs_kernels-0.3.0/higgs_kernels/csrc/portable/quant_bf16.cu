#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
#include <cstdint>
#include <limits>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAException.h>

static constexpr int kCodebookSize = 256;
static constexpr int kInDim = 2;

__device__ __forceinline__ float bf16_to_fp32(uint16_t v) {
	uint32_t u = static_cast<uint32_t>(v) << 16;
	return __uint_as_float(u);
}
__device__ __forceinline__ uint16_t fp32_to_bf16_rne(float f) {
	uint32_t x = __float_as_uint(f);
	uint32_t lsb = (x >> 16) & 1U;
	uint32_t rounding_bias = 0x00007FFFU + lsb;
	x += rounding_bias;
	return static_cast<uint16_t>(x >> 16);
}

__device__ __forceinline__ uint16_t bf16_mul(uint16_t a_bf16, uint16_t b_bf16) {
	float a = bf16_to_fp32(a_bf16);
	float b = bf16_to_fp32(b_bf16);
	return fp32_to_bf16_rne(a * b);
}

__global__ void higgs_quantize_2_256_ptr_bf16_cuda_portable_kernel(
	const uint16_t* __restrict__ x,
	const uint16_t* __restrict__ grid,
	const uint16_t* __restrict__ grid_norms,
	unsigned char* __restrict__ out,
	int64_t out_dim)
{
	__shared__ float s_grid[kCodebookSize][kInDim];
	__shared__ uint16_t s_norms_bf16[kCodebookSize];

	for (int idx = threadIdx.x; idx < kCodebookSize * kInDim; idx += blockDim.x) {
		int r = idx / kInDim;
		int c = idx % kInDim;
		s_grid[r][c] = bf16_to_fp32(grid[r * kInDim + c]);
	}
	for (int idx = threadIdx.x; idx < kCodebookSize; idx += blockDim.x) {
		s_norms_bf16[idx] = grid_norms[idx];
	}
	__syncthreads();

	int64_t row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= out_dim) return;

	const uint16_t two_bf16 = fp32_to_bf16_rne(2.0f);

	uint16_t x0_b = x[row * kInDim + 0];
	uint16_t x1_b = x[row * kInDim + 1];
	float x0 = bf16_to_fp32(x0_b);
	float x1 = bf16_to_fp32(x1_b);

	float best_score = -HIP_INF_F;
	unsigned int best_index = 0u;

	#pragma unroll 8
	for (int c = 0; c < kCodebookSize; ++c) {
		float g0 = s_grid[c][0];
		float g1 = s_grid[c][1];
		float dot_fp32 = x0 * g0 + x1 * g1;
		uint16_t dot_bf16 = fp32_to_bf16_rne(dot_fp32);

		uint16_t twice_dot_bf16 = bf16_mul(dot_bf16, two_bf16);
		float twice_dot = bf16_to_fp32(twice_dot_bf16);

		uint16_t grid_norm_bf16 = s_norms_bf16[c];
		float score = bf16_to_fp32(fp32_to_bf16_rne(twice_dot - bf16_to_fp32(grid_norm_bf16)));

		if (score > best_score) {
			best_score = score;
			best_index = static_cast<unsigned int>(c);
		}
	}

	out[row] = static_cast<unsigned char>(best_index);
}

extern "C" void higgs_quantize_2_256_ptr_bf16_cuda_portable(
	uint64_t x_ptr,
	uint64_t grid_ptr,
	uint64_t grid_norms_ptr,
	uint64_t out_ptr,
	int64_t out_dim)
{
	const uint16_t* x = reinterpret_cast<const uint16_t*>(x_ptr);
	const uint16_t* grid = reinterpret_cast<const uint16_t*>(grid_ptr);
	const uint16_t* grid_norms = reinterpret_cast<const uint16_t*>(grid_norms_ptr);
	unsigned char* out = reinterpret_cast<unsigned char*>(out_ptr);

	int threads = 256;
	int blocks = static_cast<int>((out_dim + threads - 1) / threads);

	auto stream = at::cuda::getCurrentCUDAStream();
	higgs_quantize_2_256_ptr_bf16_cuda_portable_kernel<<<blocks, threads, 0, stream>>>(x, grid, grid_norms, out, out_dim);

	C10_CUDA_KERNEL_LAUNCH_CHECK();
}