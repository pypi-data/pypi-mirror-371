#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <c10/cuda/CUDAStream.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>


#define REP_1_8(x, y, ...) \
    { constexpr int x = 1; if (y == x) {__VA_ARGS__;} } \
    { constexpr int x = 2; if (y == x) {__VA_ARGS__;} } \
    { constexpr int x = 3; if (y == x) {__VA_ARGS__;} } \
    { constexpr int x = 4; if (y == x) {__VA_ARGS__;} } \
    { constexpr int x = 5; if (y == x) {__VA_ARGS__;} } \
    { constexpr int x = 6; if (y == x) {__VA_ARGS__;} } \
    { constexpr int x = 7; if (y == x) {__VA_ARGS__;} } \
    { constexpr int x = 8; if (y == x) {__VA_ARGS__;} }


#define OP_PER_LANE 1
#define MAX_WARP_BLOCK_RATIO 4
#define MAX_SPLIT_K 32

namespace bf16_huffman_infer {

static int ceil_div(int a, int b) {
    return (a + b - 1) / b;
}


template <int batch_size>
__global__ void gemv_bf16_kernel(
    const nv_bfloat162* A, const nv_bfloat162* X, hip_bfloat16* Y,
    int M, int N
) {
    N /= 2;

    int thread_id = ((blockIdx.x * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x) * 2;

    int warp_group_size = warpSize * 2;

    int warp_group_id = thread_id / warp_group_size;
    int lane_id = thread_id % warp_group_size;

    int stride = N;
    
    if (warp_group_id * OP_PER_LANE > M) {
        return; // no work to do
    }

    const int boundry_offset = max((warp_group_id + 1) * OP_PER_LANE - M, 0);

    const std::array<nv_bfloat162, 2> *pa = (const std::array<nv_bfloat162, 2> *)&A[(warp_group_id * OP_PER_LANE - boundry_offset) * stride + lane_id];
    const std::array<nv_bfloat162, 2> *px = (const std::array<nv_bfloat162, 2> *)&X[lane_id];

    std::array<nv_bfloat162, 2> x[batch_size];
    std::array<nv_bfloat162, 2> a[OP_PER_LANE];
    float y[batch_size][OP_PER_LANE] = {};

    __syncwarp();

    for (int count = 0, n_iter = N / warp_group_size; count < n_iter; count += 1) {
        #pragma unroll
        for (int i = 0; i < batch_size; i++) {
            x[i] = px[i * (N * 2 / (sizeof(px[0]) / sizeof(hip_bfloat16)))];
        }
        const std::array<nv_bfloat162, 2> *npa = pa;
        #pragma unroll
        for (int i = 0; i < OP_PER_LANE; i++) {
            a[i] = *npa;
            npa += stride / 2;
        }
        pa += warpSize;
        px += warpSize;


        float2 v0[batch_size], v1[batch_size];
        #pragma unroll
        for (int i = 0; i < batch_size; i++) {
            v0[i] = __bfloat1622float2(x[i][0]);
            v1[i] = __bfloat1622float2(x[i][1]);
        }
        #pragma unroll
        for (int i = 0; i < OP_PER_LANE; i++) {
            auto u0 = __bfloat1622float2(a[i][0]);
            auto u1 = __bfloat1622float2(a[i][1]);
            for (int b = 0; b < batch_size; b++) {
                y[b][i] += (u0.x * v0[b].x + u0.y * v0[b].y) + (u1.x * v1[b].x + u1.y * v1[b].y);
            }
        }
    }

    // warp reduce on y
    __syncwarp();
    #pragma unroll
    for (int b = 0; b < batch_size; b++) {
        #pragma unroll
        for (int i = 0; i < OP_PER_LANE; i++) {
            #pragma unroll
            for (int j = warpSize / 2; j > 0; j /= 2) {
                y[b][i] += __shfl_down_sync(0xFFFFFFFF, y[b][i], j);
            }
        }
    }

    // __syncthreads();
    __syncwarp();

    if (lane_id == 0) {
        #pragma unroll
        for (int b = 0; b < batch_size; b++) {
            #pragma unroll
            for (int i = 0; i < OP_PER_LANE; i++) {
                Y[(warp_group_id * OP_PER_LANE - boundry_offset) + i] = __float2bfloat16(y[b][i]);
            }
            Y += M;
        }
    }
}


void gemv_bf16(
    const torch::Tensor &A,
    const torch::Tensor &X,
    torch::Tensor &Y
) {
    int M = A.size(0);
    int N = A.size(1);

    int num_warps_per_block = 2;
    auto block_size = dim3(32, num_warps_per_block, 1);
    int grid_size = ceil_div(M, OP_PER_LANE * num_warps_per_block);

    auto stream = c10::cuda::getCurrentCUDAStream(A.device().index()).stream();

    int batch_size = X.size(0);
    TORCH_CHECK_LE(batch_size, 8);

    REP_1_8(
        b, batch_size,
        gemv_bf16_kernel<b><<<grid_size, block_size, 0, stream>>>(
            static_cast<const nv_bfloat162*>(A.const_data_ptr()),
            static_cast<const nv_bfloat162*>(X.const_data_ptr()),
            static_cast<hip_bfloat16*>(Y.mutable_data_ptr()),
            M, N
        )
    );
}


struct LUT {
    uint8_t LUT1[256];
    uint8_t LUT2[256];
    uint8_t LUT3[256];
    uint8_t LUT4[256];
    uint8_t code_lengths[256];
};


struct decoder{
    union {
        uint64_t data;
        uchar4 v;
    } state{0};
    uint8_t remaining_bits = 0;

    __device__ __inline__ uint8_t decode_symbol(
        const uint32_t* &pae, int warp_group_size,
        const uint8_t* LUT1, const uint8_t* LUT2, const uint8_t* LUT3, const uint8_t* LUT4,
        const uint8_t* code_lengths
    ) {
        uint8_t symbol;

        if (remaining_bits < 32) {
            state.data |= uint64_t(*pae) << remaining_bits;
            pae += warp_group_size;
            remaining_bits += 32;
        }
        
        if ((symbol = LUT1[state.v.x]) != 255);
        else if ((symbol = LUT2[state.v.y]) != 255);
        else if ((symbol = LUT3[state.v.z]) != 255);
        else if ((symbol = LUT4[state.v.w]) != 255);
        // else assert(0);
        auto bitoffset = code_lengths[symbol];
        state.data >>= bitoffset;
        remaining_bits -= bitoffset;

        return symbol;
    }

    __device__ __inline__ uint8_t decode_symbol2(
        const uint32_t* &pae, int warp_group_size, const LUT *lut
    ) {
        uint8_t symbol;

        if (remaining_bits < 32) {
            // TODO: *pae is interleaved load, fix it
            state.data |= uint64_t(*pae) << remaining_bits;
            pae += warp_group_size;
            remaining_bits += 32;
        }
        
        if ((symbol = lut->LUT1[state.v.x]) != 255);
        else if ((symbol = lut->LUT2[state.v.y]) != 255);
        else if ((symbol = lut->LUT3[state.v.z]) != 255);
        else if ((symbol = lut->LUT4[state.v.w]) != 255);
        // else assert(0);
        auto bitoffset = lut->code_lengths[symbol];
        state.data >>= bitoffset;
        remaining_bits -= bitoffset;

        return symbol;
    }
};


template <int width> struct vector_type {};
template <> struct vector_type<1> { using type = uint1; };
template <> struct vector_type<2> { using type = uint2; };
template <> struct vector_type<4> { using type = uint4; };

template <typename T, int width>
union vec {
    using vector_type = typename vector_type<width>::type;
    vector_type data;
    T value[width];

    __device__ __inline__ vec<T, width>& operator=(const vec<T, width>& other) {
        data = other.data;
        return *this;
    }

    __device__ __inline__ vec<T, width>& operator=(vec<T, width>&& other) {
        data = other.data;
        return *this;
    }

    template <typename I>
    __device__ __inline__ T operator[](I index) {
        return value[index];
    }
};


template <int batch_size>
__global__ void
gemv_bf16_huffman_kernel(
    const uchar4* A_rem, const uint32_t* A_exp, const nv_bfloat162* X, hip_bfloat16* Y,
    const uint32_t* offsets,
    const uint8_t* LUT1, const uint8_t* LUT2, const uint8_t* LUT3, const uint8_t* LUT4,
    const uint8_t* code_lengths,
    int M, int N, int split_k
) {
    __shared__ LUT sh_LUT;

    ((uint64_t*)sh_LUT.LUT1)[threadIdx.x] = ((const uint64_t*)LUT1)[threadIdx.x];
    ((uint64_t*)sh_LUT.LUT2)[threadIdx.x] = ((const uint64_t*)LUT2)[threadIdx.x];
    ((uint64_t*)sh_LUT.LUT3)[threadIdx.x] = ((const uint64_t*)LUT3)[threadIdx.x];
    ((uint64_t*)sh_LUT.LUT4)[threadIdx.x] = ((const uint64_t*)LUT4)[threadIdx.x];
    ((uint64_t*)sh_LUT.code_lengths)[threadIdx.x] = ((const uint64_t*)code_lengths)[threadIdx.x];

    __shared__ struct {
        float y[MAX_WARP_BLOCK_RATIO][batch_size][OP_PER_LANE][MAX_SPLIT_K];
        int count[MAX_WARP_BLOCK_RATIO];
    } tmp;

    if (threadIdx.x == 0 && threadIdx.y == 0) {
        tmp.count[threadIdx.z] = 0;
    }
    assert(blockDim.z <= MAX_WARP_BLOCK_RATIO);
    assert(split_k <= MAX_SPLIT_K);

    __syncthreads();

    assert(blockDim.x == warpSize);

    int warp_group_id = blockIdx.x * blockDim.z + threadIdx.z;
    int lane_id = threadIdx.x;
    int thread_id = warp_group_id * blockDim.x + threadIdx.x;

    if (warp_group_id * OP_PER_LANE >= M) {
        return; // no work to do
    }

    float y[batch_size][OP_PER_LANE] = {};

    int k = threadIdx.y;
    // int k = warp_group_id / (M / OP_PER_LANE);
    // warp_group_id %= (M / OP_PER_LANE);

    A_rem += M * N / sizeof(A_rem[0]) * k;
    X += N / (sizeof(X[0]) / sizeof(hip_bfloat16)) * k;
    offsets += M * k;

    int stride = N / 4;

    // const vec<nv_bfloat162, 2> *px = &X[lane_id];
    const nv_bfloat162 *px = &X[lane_id];
    const uchar4 *par = &A_rem[(warp_group_id * OP_PER_LANE) * stride + lane_id];

    const uint32_t *pae0 = &A_exp[offsets[warp_group_id] + lane_id + 0];
    const uint32_t *pae1 = &A_exp[offsets[warp_group_id] + lane_id + warpSize];

    // vec<nv_bfloat162, 2> x[batch_size];
    nv_bfloat162 x[batch_size][2];
    uchar4 ar[OP_PER_LANE];
    uchar4 ae[OP_PER_LANE];

    decoder dec0;
    decoder dec1;

    __syncwarp();

    for (int count = 0, n_iter = N / (4 * warpSize); count < n_iter; count += 1) {
        #pragma unroll
        for (int i = 0; i < batch_size; i++) {
            // NOTE: it will not work as expected: vector load 64bit, if using array<nv_bfloat162,2>
            // instead, it load 2 32bits load, with interleaved layout, which is much slower
            // x[i] = px[i * (split_k * N / (sizeof(px[0]) / sizeof(hip_bfloat16)))];
            x[i][0] = px[i * (split_k * N / (sizeof(px[0]) / sizeof(hip_bfloat16))) + 0];
            x[i][1] = px[i * (split_k * N / (sizeof(px[0]) / sizeof(hip_bfloat16))) + warpSize];
        }
        const uchar4 *npar = par;
        #pragma unroll
        for (int i = 0; i < OP_PER_LANE; i++) {
            ar[i] = *npar;
            npar += stride;
        }
        par += warpSize;
        px += warpSize * 2;

        #pragma unroll
        for (int i = 0; i < OP_PER_LANE; i++) {
            ae[i].x = dec0.decode_symbol2(pae0, warpSize * 2, &sh_LUT);
            ae[i].z = dec1.decode_symbol2(pae1, warpSize * 2, &sh_LUT);
            ae[i].y = dec0.decode_symbol2(pae0, warpSize * 2, &sh_LUT);
            ae[i].w = dec1.decode_symbol2(pae1, warpSize * 2, &sh_LUT);
        }

        // __syncwarp();

        float2 v0[batch_size], v1[batch_size];
        #pragma unroll
        for (int i = 0; i < batch_size; i++) {
            v0[i] = __bfloat1622float2(x[i][0]);
            v1[i] = __bfloat1622float2(x[i][1]);
        }

        // auto v0 = __bfloat1622float2(x[0]);
        // auto v1 = __bfloat1622float2(x[1]);

        #pragma unroll
        for (int i = 0; i < OP_PER_LANE; i++) {
            uint32_t rem0 = (uint32_t(ar[i].y) << 16) | ar[i].x;
            uint32_t rem1 = (uint32_t(ar[i].w) << 16) | ar[i].z;
            uint32_t exp0 = (uint32_t(ae[i].y) << 16) | ae[i].x;
            uint32_t exp1 = (uint32_t(ae[i].w) << 16) | ae[i].z;
            union {
                uint32_t _bits;
                nv_bfloat162 u;
            } bf160{((rem0 << 8) & 0x80008000) | (rem0 & 0x007F007F) | (exp0 << 7)};
            union {
                uint32_t _bits;
                nv_bfloat162 u;
            } bf161{((rem1 << 8) & 0x80008000) | (rem1 & 0x007F007F) | (exp1 << 7)};
            auto u0 = __bfloat1622float2(bf160.u);
            auto u1 = __bfloat1622float2(bf161.u);
            #pragma unroll
            for (int j = 0; j < batch_size; j++) {
                y[j][i] += (u0.x * v0[j].x + u0.y * v0[j].y) + (u1.x * v1[j].x + u1.y * v1[j].y);
            }
        }
    }

    
    // warp reduce on y
    __syncwarp();
    #pragma unroll
    for (int b = 0; b < batch_size; b++) {
        #pragma unroll
        for (int i = 0; i < OP_PER_LANE; i++) {
            #pragma unroll
            for (int j = warpSize / 2; j > 0; j /= 2) {
                y[b][i] += __shfl_down_sync(0xFFFFFFFF, y[b][i], j);
            }
        }
    }

    // __syncthreads();
    __syncwarp();

    if (lane_id == 0) {
        #pragma unroll
        for (int b = 0; b < batch_size; b++) {
            #pragma unroll
            for (int i = 0; i < OP_PER_LANE; i++) {
                // Y[(warp_group_id * OP_PER_LANE) + i] = __float2bfloat16(y[b][i]);
                // atomicAdd(&Y[(warp_group_id * OP_PER_LANE) + i], __float2bfloat16(y[b][i]));
                // atomicAdd(&Y[(warp_group_id * OP_PER_LANE) + i], y[b][i]);
                tmp.y[threadIdx.z][b][i][k] = y[b][i];
            }
            // Y += M;
        }
        // Y -= M * batch_size; // reset Y pointer to the start of the batch

        int res = atomicAdd_block(&tmp.count[threadIdx.z], 1);
        if (res == split_k - 1) {
            // last thread in the block, write back the results
            #pragma unroll
            for (int b = 0; b < batch_size; b++) {
                #pragma unroll
                for (int i = 0; i < OP_PER_LANE; i++) {
                    float y = 0.0;
                    for (int j = 0; j < split_k; j++) {
                        y += tmp.y[threadIdx.z][b][i][j];
                    }
                    Y[(warp_group_id * OP_PER_LANE) + i] = __float2bfloat16(y);
                }
                Y += M;
            }
            Y -= M * batch_size; // reset Y pointer to the start of the batch
        }
    }
}


void gemv_bf16_huffman(
    const torch::Tensor &A_rem,
    const torch::Tensor &A_exp,
    const torch::Tensor &X,
    torch::Tensor &Y,
    const torch::Tensor &offsets,
    const torch::Tensor &LUT1,
    const torch::Tensor &LUT2,
    const torch::Tensor &LUT3,
    const torch::Tensor &LUT4,
    const torch::Tensor &code_lengths
) {
    int split_k = A_rem.size(0);
    int M = A_rem.size(1);
    int N = A_rem.size(2);

    hipDeviceProp_t attr;
    TORCH_CHECK(hipGetDeviceProperties(&attr, A_rem.device().index()) == hipSuccess);
    int num_warps_per_block = attr.maxThreadsPerMultiProcessor / 32 / attr.maxBlocksPerMultiProcessor;
    num_warps_per_block = ceil_div(num_warps_per_block, split_k);

    auto block_size = dim3(32, split_k, num_warps_per_block);
    auto grid_size = dim3(ceil_div(M, OP_PER_LANE * num_warps_per_block), 1, 1);

    auto stream = c10::cuda::getCurrentCUDAStream(A_rem.device().index()).stream();

    int batch_size = X.size(0);
    TORCH_CHECK_LE(batch_size, 8);

    REP_1_8(
        b, batch_size,
        gemv_bf16_huffman_kernel<b><<<grid_size, block_size, 0, stream>>>(
            static_cast<const uchar4*>(A_rem.const_data_ptr()),
            static_cast<const uint32_t*>(A_exp.const_data_ptr()),
            static_cast<const nv_bfloat162*>(X.const_data_ptr()),
            static_cast<hip_bfloat16*>(Y.mutable_data_ptr()),
            static_cast<const uint32_t*>(offsets.const_data_ptr()),
            static_cast<const uint8_t*>(LUT1.const_data_ptr()),
            static_cast<const uint8_t*>(LUT2.const_data_ptr()),
            static_cast<const uint8_t*>(LUT3.const_data_ptr()),
            static_cast<const uint8_t*>(LUT4.const_data_ptr()),
            static_cast<const uint8_t*>(code_lengths.const_data_ptr()),
            M, N, split_k
        )
    );
}


__global__ void huffman_decode_kernel(
    const uchar2* A_rem, const uint32_t* A_exp, nv_bfloat162* Y,
    const uint32_t* offsets,
    const uint8_t* LUT1, const uint8_t* LUT2, const uint8_t* LUT3, const uint8_t* LUT4,
    const uint8_t* code_lengths,
    int M, int N, int split_k
) {
    __shared__ LUT sh_LUT;

    ((uint64_t*)sh_LUT.LUT1)[threadIdx.x] = ((const uint64_t*)LUT1)[threadIdx.x];
    ((uint64_t*)sh_LUT.LUT2)[threadIdx.x] = ((const uint64_t*)LUT2)[threadIdx.x];
    ((uint64_t*)sh_LUT.LUT3)[threadIdx.x] = ((const uint64_t*)LUT3)[threadIdx.x];
    ((uint64_t*)sh_LUT.LUT4)[threadIdx.x] = ((const uint64_t*)LUT4)[threadIdx.x];
    ((uint64_t*)sh_LUT.code_lengths)[threadIdx.x] = ((const uint64_t*)code_lengths)[threadIdx.x];

    __syncthreads();

    int thread_id = ((blockIdx.x * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x) * 2;

    int warp_group_size = warpSize * 2;

    int warp_group_id = thread_id / warp_group_size;
    int lane_id = thread_id % warp_group_size;

    if (warp_group_id * OP_PER_LANE > M) {
        return; // no work to do
    }

    for (int k = 0; k < split_k; k++) {
        int stride = N / 2;

        const uchar4 *par = (const uchar4 *)&A_rem[(warp_group_id * OP_PER_LANE) * stride + lane_id];
        const uint32_t *pae = &A_exp[offsets[warp_group_id] + lane_id / 2];
        const uint32_t *pae2 = &A_exp[offsets[warp_group_id] + lane_id / 2 + warpSize];

        uchar4 ar[OP_PER_LANE];
        uchar4 ae[OP_PER_LANE];

        decoder dec;
        decoder dec2;

        __syncwarp();

        for (int count = 0, n_iter = N / (2 * warp_group_size); count < n_iter; count += 1) {
            const uchar4 *npar = par;
            #pragma unroll
            for (int i = 0; i < OP_PER_LANE; i++) {
                ar[i] = *npar;
                npar += stride / 2;
            }
            par += warpSize;

            #pragma unroll
            for (int i = 0; i < OP_PER_LANE; i++) {
                ae[i].x = dec.decode_symbol2(pae, warp_group_size, &sh_LUT);
                ae[i].z = dec2.decode_symbol2(pae2, warp_group_size, &sh_LUT);
                ae[i].y = dec.decode_symbol2(pae, warp_group_size, &sh_LUT);
                ae[i].w = dec2.decode_symbol2(pae2, warp_group_size, &sh_LUT);
            }

            // __syncwarp();

            #pragma unroll
            for (int i = 0; i < OP_PER_LANE; i++) {
                uint32_t rem0 = (uint32_t(ar[i].y) << 16) | ar[i].x;
                uint32_t rem1 = (uint32_t(ar[i].w) << 16) | ar[i].z;
                uint32_t exp0 = (uint32_t(ae[i].y) << 16) | ae[i].x;
                uint32_t exp1 = (uint32_t(ae[i].w) << 16) | ae[i].z;
                union {
                    uint32_t _bits;
                    nv_bfloat162 u;
                } bf160{((rem0 << 8) & 0x80008000) | (rem0 & 0x007F007F) | (exp0 << 7)};
                union {
                    uint32_t _bits;
                    nv_bfloat162 u;
                } bf161{((rem1 << 8) & 0x80008000) | (rem1 & 0x007F007F) | (exp1 << 7)};
                Y[(warp_group_id * OP_PER_LANE + i) * N / 2 + count * warp_group_size + lane_id / 2] = bf160.u;
                Y[(warp_group_id * OP_PER_LANE + i) * N / 2 + count * warp_group_size + lane_id / 2 + warpSize] = bf161.u;
            }
        }
        
        {
            // handle split k
            int num_warp_groups = blockDim.y * gridDim.x;
            int offsets_stride = num_warp_groups;
            // printf("%d\n", offsets_stride);

            // N /= split_k;
            A_rem += M * N / sizeof(A_rem[0]);
            Y += M * N / (sizeof(Y[0]) / sizeof(hip_bfloat16));
            offsets += offsets_stride;
        }
    }
}


void huffman_decode(
    const torch::Tensor &A_rem,
    const torch::Tensor &A_exp,
    torch::Tensor &Y,
    const torch::Tensor &offsets,
    const torch::Tensor &LUT1,
    const torch::Tensor &LUT2,
    const torch::Tensor &LUT3,
    const torch::Tensor &LUT4,
    const torch::Tensor &code_lengths
) {
    int split_k = A_rem.size(0);
    int M = A_rem.size(1);
    int N = A_rem.size(2);

    int num_warps_per_block = 4; // TODO: If 3 will crash randomly
    auto block_size = dim3(32, num_warps_per_block, 1);
    auto grid_size = dim3(ceil_div(M, OP_PER_LANE * num_warps_per_block), 1, 1);

    auto stream = c10::cuda::getCurrentCUDAStream(A_rem.device().index()).stream();

    huffman_decode_kernel<<<grid_size, block_size, 0, stream>>>(
        static_cast<const uchar2*>(A_rem.const_data_ptr()),
        static_cast<const uint32_t*>(A_exp.const_data_ptr()),
        static_cast<nv_bfloat162*>(Y.mutable_data_ptr()),
        static_cast<const uint32_t*>(offsets.const_data_ptr()),
        static_cast<const uint8_t*>(LUT1.const_data_ptr()),
        static_cast<const uint8_t*>(LUT2.const_data_ptr()),
        static_cast<const uint8_t*>(LUT3.const_data_ptr()),
        static_cast<const uint8_t*>(LUT4.const_data_ptr()),
        static_cast<const uint8_t*>(code_lengths.const_data_ptr()),
        M, N, split_k
    );
}


__global__ void huffman_encode_kernel(
    const uint8_t *data,
    uint32_t data_length,
    int num_data,
    const char* LUT,
    char *output,
    uint32_t output_lengths[]
) {
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (thread_id >= num_data) return;

    uint32_t output_count = 0;
    for (int i = 0; i < data_length; i++) {
        const char *p = &LUT[data[thread_id * data_length + i] * 32];
        for (char ch = *p++, count = 0; ch != '\0' && count < 32; count++, ch = *p++) {
            output[thread_id * data_length * 32 + output_count] = ch;
            output_count++;
        }
    }
    output_lengths[thread_id] = output_count;
}


void huffman_encode(
    const torch::Tensor &data,
    const torch::Tensor &LUT,
    torch::Tensor &output,
    torch::Tensor &output_lengths
) { 
    int num_data = data.size(0);
    int data_lengths = data.size(1);
    int block_size = 32;
    int grid_size = ceil_div(num_data, block_size);
    auto stream = c10::cuda::getCurrentCUDAStream(data.device().index()).stream();
    huffman_encode_kernel<<<grid_size, block_size, 0, stream>>>(
        static_cast<const uint8_t*>(data.const_data_ptr()),
        data_lengths,
        num_data,
        static_cast<const char*>(LUT.const_data_ptr()),
        static_cast<char*>(output.mutable_data_ptr()),
        static_cast<uint32_t*>(output_lengths.mutable_data_ptr())
    );
}

TORCH_LIBRARY_IMPL(bf16_huffman_infer, CUDA, m) {
    m.impl("gemv_bf16", &gemv_bf16);
    m.impl("gemv_bf16_huffman", &gemv_bf16_huffman);
    m.impl("huffman_encode", &huffman_encode);
    m.impl("huffman_decode", &huffman_decode);
}

}
