#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <ATen/cuda/HIPContext.h> // For getCurrentCUDAStream

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")

inline void gpuCheck(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
        const char* errName = hipGetErrorName(code);
        const char* errString = hipGetErrorString(code);
        TORCH_CHECK(false, "CUDA error: ", errName, " ", errString, " at ", file, ":", line);
    }
  }
#define CUDA_CHECK(ans) { gpuCheck((ans), __FILE__, __LINE__); }

// Optimized kernel with improved memory access patterns
__global__ void float_round_kernel_inplace(float* input,
                                           int N,
                                           float max_exp,
                                           float min_exp,
                                           int mantissa_upper_bound,
                                           float mantissa_scale,
                                           float inv_mantissa_scale) {
    // Use vectorized loads for better memory coalescing
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    // Process multiple elements per thread to improve memory bandwidth utilization
    for (int idx = tid; idx < N; idx += stride) {
        float x_val = input[idx];

        // Early exit for zero values (reduces unnecessary computation)
        if (x_val == 0.0f) continue;

        // Use fast math intrinsics for better performance
        const float s = copysignf(1.0f, x_val);
        const float x_abs = fabsf(x_val);

        // Use fast log2 and exp2 intrinsics
        const float exponent_floor = log2f(x_abs);
        float exponent = fmaxf(fminf(exponent_floor, max_exp), min_exp);
        float exp2_val = exp2f(exponent);

        // Optimize division with reciprocal multiplication
        float scaled = fmaf(x_abs, __frcp_rn(exp2_val), 0.0f);
        scaled = fmaxf(scaled, 1.0f);

        // Use FMA for better instruction fusion
        const float mantissa_unrounded = fmaf(scaled - 1.0f, mantissa_scale, 0.0f);
        const int mantissa = __float2int_rn(mantissa_unrounded);

        // Branchless overflow handling with predicated execution
        const bool overflow = mantissa >= mantissa_upper_bound;
        const float exponent_overflow = fmaxf(fminf(fmaf(exponent, 1.0f, 1.0f), max_exp), min_exp);
        const float exp2_val_overflow = exp2f(exponent_overflow);

        // Select final values without branches using predication
        const float final_exp2 = overflow ? exp2_val_overflow : exp2_val;
        const int final_mantissa = overflow ? 0 : mantissa;

        // Use FMA for final computation
        const float fraction = static_cast<float>(final_mantissa) * inv_mantissa_scale;
        input[idx] = fmaf(fmaf(fraction, final_exp2, final_exp2), s, 0.0f);
    }
}

// Vectorized kernel using float4 for maximum memory bandwidth
__global__ void float_round_kernel_vectorized(float4* input_vec,
                                             int N_vec,
                                             float max_exp,
                                             float min_exp,
                                             int mantissa_upper_bound,
                                             float mantissa_scale,
                                             float inv_mantissa_scale) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    // Process float4 elements (4 floats per thread)
    for (int idx = tid; idx < N_vec; idx += stride) {
        float4 vec = input_vec[idx];

        // Process each component of the float4 vector
        #pragma unroll
        for (int i = 0; i < 4; ++i) {
            float* x_ptr = reinterpret_cast<float*>(&vec) + i;
            float x_val = *x_ptr;

            if (x_val == 0.0f) continue;

            // Use fast math intrinsics
            const float s = copysignf(1.0f, x_val);
            const float x_abs = fabsf(x_val);
            const float exponent_floor = log2f(x_abs);
            float exponent = fmaxf(fminf(exponent_floor, max_exp), min_exp);
            float exp2_val = exp2f(exponent);

            // Optimized computation with FMA
            float scaled = fmaf(x_abs, __frcp_rn(exp2_val), 0.0f);
            scaled = fmaxf(scaled, 1.0f);

            const float mantissa_unrounded = fmaf(scaled - 1.0f, mantissa_scale, 0.0f);
            const int mantissa = __float2int_rn(mantissa_unrounded);

            const bool overflow = mantissa >= mantissa_upper_bound;
            const float exponent_overflow = fmaxf(fminf(fmaf(exponent, 1.0f, 1.0f), max_exp), min_exp);
            const float exp2_val_overflow = exp2f(exponent_overflow);

            const float final_exp2 = overflow ? exp2_val_overflow : exp2_val;
            const int final_mantissa = overflow ? 0 : mantissa;

            const float fraction = static_cast<float>(final_mantissa) * inv_mantissa_scale;
            *x_ptr = fmaf(fmaf(fraction, final_exp2, final_exp2), s, 0.0f);
        }

        // Store the processed float4 vector
        input_vec[idx] = vec;
    }
}

// Shared memory optimized kernel for better cache utilization
__global__ void float_round_kernel_shared(float* input,
                                         int N,
                                         float max_exp,
                                         float min_exp,
                                         int mantissa_upper_bound,
                                         float mantissa_scale,
                                         float inv_mantissa_scale) {
    __shared__ float shared_data[1024]; // Shared memory buffer

    const int tid = threadIdx.x;

    for (int base_idx = blockIdx.x * blockDim.x; base_idx < N; base_idx += blockDim.x * gridDim.x) {
        int idx = base_idx + tid;

        // Load data into shared memory with coalesced access
        if (idx < N) {
            shared_data[tid] = input[idx];
        } else {
            shared_data[tid] = 0.0f;
        }

        __syncthreads();

        // Process data from shared memory
        if (idx < N) {
            float x_val = shared_data[tid];

            if (x_val != 0.0f) {
                // Use fast math intrinsics
                const float s = copysignf(1.0f, x_val);
                const float x_abs = fabsf(x_val);
                const float exponent_floor = log2f(x_abs);
                float exponent = fmaxf(fminf(exponent_floor, max_exp), min_exp);
                float exp2_val = exp2f(exponent);

                // Optimized computation
                float scaled = fmaf(x_abs, __frcp_rn(exp2_val), 0.0f);
                scaled = fmaxf(scaled, 1.0f);

                const float mantissa_unrounded = fmaf(scaled - 1.0f, mantissa_scale, 0.0f);
                const int mantissa = __float2int_rn(mantissa_unrounded);

                const bool overflow = mantissa >= mantissa_upper_bound;
                const float exponent_overflow = fmaxf(fminf(fmaf(exponent, 1.0f, 1.0f), max_exp), min_exp);
                const float exp2_val_overflow = exp2f(exponent_overflow);

                const float final_exp2 = overflow ? exp2_val_overflow : exp2_val;
                const int final_mantissa = overflow ? 0 : mantissa;

                const float fraction = static_cast<float>(final_mantissa) * inv_mantissa_scale;
                shared_data[tid] = fmaf(fmaf(fraction, final_exp2, final_exp2), s, 0.0f);
            }
        }

        __syncthreads();

        // Store back to global memory with coalesced access
        if (idx < N) {
            input[idx] = shared_data[tid];
        }
    }
}

// Function that launches the optimized kernel
torch::Tensor float_round_cuda_inplace(torch::Tensor input, int exponent_bits, int mantissa_bits, int bias) {
    CHECK_CUDA(input);

    int numel = input.numel();
    if (numel == 0) return input;

    // Precompute constants
    int max_exp_val = (1 << exponent_bits) - 1 - bias;
    float max_exp = static_cast<float>(max_exp_val);
    float min_exp = static_cast<float>(-bias);
    int mantissa_upper_bound = 1 << mantissa_bits;
    float mantissa_scale = static_cast<float>(mantissa_upper_bound);
    float inv_mantissa_scale = 1.0f / mantissa_scale;

    float* input_ptr = input.data_ptr<float>();

    // Optimize block and grid size for better occupancy
    int device_id = input.device().index();
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device_id);

    // Calculate optimal block size based on register usage and shared memory
    int threads = 256; // Reduced from 1024 to improve occupancy
    int blocks = (numel + threads - 1) / threads;

    // Ensure we don't exceed maximum blocks per SM
    int max_blocks_per_sm = prop.maxBlocksPerMultiProcessor;
    int max_blocks = prop.multiProcessorCount * max_blocks_per_sm;
    blocks = min(blocks, max_blocks);

    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    // Choose kernel based on input size and optimization strategy
    if (numel >= 1000000) {
        // For large inputs, use vectorized kernel if possible
        if (numel % 4 == 0) {
            float4* input_vec = reinterpret_cast<float4*>(input_ptr);
            int N_vec = numel / 4;
            float_round_kernel_vectorized<<<blocks, threads, 0, stream>>>(
                input_vec, N_vec, max_exp, min_exp,
                mantissa_upper_bound, mantissa_scale, inv_mantissa_scale
            );
        } else {
            // Use shared memory kernel for better cache utilization
            float_round_kernel_shared<<<blocks, threads, 0, stream>>>(
                input_ptr, numel, max_exp, min_exp,
                mantissa_upper_bound, mantissa_scale, inv_mantissa_scale
            );
        }
    } else {
        // For smaller inputs, use optimized kernel
        float_round_kernel_inplace<<<blocks, threads, 0, stream>>>(
            input_ptr, numel, max_exp, min_exp,
            mantissa_upper_bound, mantissa_scale, inv_mantissa_scale
        );
    }

    CUDA_CHECK(hipGetLastError());

    return input;
}